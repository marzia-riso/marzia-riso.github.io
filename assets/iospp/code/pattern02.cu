#include "hip/hip_runtime.h"
#include <nodes.cuh>

struct Params0Input {
  float angle;
  float2 translation;
  float2 space;
  float thickness;
};

struct Params1Input {
  float radius;
  float inner;
};

struct Params2Input {
  float radius;
  float inner;
};

struct Consts0Input {
  float double_pi;
  float pi;
};

struct Consts1Input {
  float2 two;
  float2 one;
};

struct Consts2Input {
  float2 two;
  float2 min_one;
  float2 half_one;
  float2 zero;
};

struct Consts3Input {
  float levelid0;
  float levelid1;
  float levelid2;
};

struct LevelsOutput {
  Level1Output level1;
  Level2Output level2;
  Level3Output level3;
};
__device__ __forceinline__ LevelsOutput LevelsImpl(PositionInput position, Params0Input params0, Params1Input params1, Params2Input params2) {
  Consts0Input consts0 = Consts0Input{6.28, 3.14};
  Consts1Input consts1 = Consts1Input{float2{2.0, 2.0}, float2{1.0, 1.0}};
  Consts2Input consts2 = Consts2Input{float2{2.0, 2.0}, float2{-1.0, -1.0}, float2{0.5, 0.5}, float2{0.0, 0.0}};
  Consts3Input consts3 = Consts3Input{0.0, 1.0, 2.0};
  NormalizeOut normalize0 = NormalizeImpl(NormalizeIn{params0.angle, consts0.double_pi, consts0.pi});
  Normalize2fOut normalize2f0 = Normalize2fImpl(Normalize2fIn{params0.translation, consts1.two, consts1.one});
  RotationOut rotation0 = RotationImpl(RotationIn{position.position, normalize0.value});
  TranslationOut translation0 = TranslationImpl(TranslationIn{rotation0.position, normalize2f0.value});
  RepetitionOut repetition0 = RepetitionImpl(RepetitionIn{translation0.position, params0.space});
  GridOut grid0 = GridImpl(GridIn{repetition0.position, params0.thickness});
  CircleOut circle0 = CircleImpl(CircleIn{repetition0.position, params1.radius, params1.inner});
  ScaffoldIdOut scaffoldid0 = ScaffoldIdImpl(ScaffoldIdIn{grid0.distance, consts3.levelid0, circle0.distance, consts3.levelid1, position.view_scaffold});
  Level1Output level1 = Level1Output{grid0.distance, consts3.levelid0};
  Level2Output level2 = Level2Output{scaffoldid0.distance, scaffoldid0.levelid};
  Combine2Out combine20 = Combine2Impl(Combine2In{params0.translation, consts2.two, consts2.min_one, params0.space, consts2.half_one, consts2.zero});
  TranslationOut translation1 = TranslationImpl(TranslationIn{rotation0.position, combine20.value});
  RepetitionOut repetition1 = RepetitionImpl(RepetitionIn{translation1.position, params0.space});
  CircleOut circle2 = CircleImpl(CircleIn{repetition1.position, params2.radius, params2.inner});
  OverIdOut overid2 = OverIdImpl(OverIdIn{scaffoldid0.distance, scaffoldid0.levelid, circle2.distance, consts3.levelid2});
  Level3Output level3 = Level3Output{overid2.distance, overid2.levelid};
  return LevelsOutput{level1, level2, level3};
}

