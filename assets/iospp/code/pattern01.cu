#include "hip/hip_runtime.h"
#include <nodes.cuh>

struct Params0Input {
  float angle;
  float2 translation;
  float2 space;
  float thickness;
};

struct Params1Input {
  float radius;
  float inner;
};

struct Params2Input {
  float radius;
  float inner;
};

struct Consts0Input {
  float double_pi;
  float pi;
};

struct Consts1Input {
  float2 two;
  float2 one;
};

struct Consts2Input {
  float levelid0;
  float levelid1;
  float levelid2;
};

struct LevelsOutput {
  Level1Output level1;
  Level2Output level2;
  Level3Output level3;
};
__device__ __forceinline__ LevelsOutput LevelsImpl(PositionInput position, Params0Input params0, Params1Input params1, Params2Input params2) {
  Consts0Input consts0 = Consts0Input{6.28, 3.14};
  Consts1Input consts1 = Consts1Input{float2{2.0, 2.0}, float2{1.0, 1.0}};
  Consts2Input consts2 = Consts2Input{0.0, 1.0, 2.0};
  NormalizeOut normalize0 = NormalizeImpl(NormalizeIn{params0.angle, consts0.double_pi, consts0.pi});
  Normalize2fOut normalize2f0 = Normalize2fImpl(Normalize2fIn{params0.translation, consts1.two, consts1.one});
  RotationOut rotation0 = RotationImpl(RotationIn{position.position, normalize0.value});
  TranslationOut translation0 = TranslationImpl(TranslationIn{rotation0.position, normalize2f0.value});
  RepetitionOut repetition0 = RepetitionImpl(RepetitionIn{translation0.position, params0.space});
  GridOut grid0 = GridImpl(GridIn{repetition0.position, params0.thickness});
  CircleOut circle0 = CircleImpl(CircleIn{repetition0.position, params1.radius, params1.inner});
  CircleOut circle1 = CircleImpl(CircleIn{repetition0.position, params2.radius, params2.inner});
  ScaffoldIdOut scaffoldid0 = ScaffoldIdImpl(ScaffoldIdIn{grid0.distance, consts2.levelid0, circle0.distance, consts2.levelid1, position.view_scaffold});
  Level1Output level1 = Level1Output{grid0.distance, consts2.levelid0};
  Level2Output level2 = Level2Output{scaffoldid0.distance, scaffoldid0.levelid};
  OverIdOut overid1 = OverIdImpl(OverIdIn{scaffoldid0.distance, scaffoldid0.levelid, circle1.distance, consts2.levelid2});
  Level3Output level3 = Level3Output{overid1.distance, overid1.levelid};
  return LevelsOutput{level1, level2, level3};
}

