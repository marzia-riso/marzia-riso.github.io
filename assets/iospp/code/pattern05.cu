#include "hip/hip_runtime.h"
#include <nodes.cuh>

struct Params0Input {
  float scale;
  float theta;
  float2 trans;
  float thickness;
};

struct Params1Input {
  float thickness;
  float rotation;
};

struct Consts0Input {
  float double_pi;
  float pi;
};

struct Consts1Input {
  float2 two;
  float2 one;
};

struct Consts2Input {
  float2 space;
};

struct Consts3Input {
  float levelid0;
  float levelid1;
};

struct LevelsOutput {
  Level1Output level1;
  Level2Output level2;
};
__device__ __forceinline__ LevelsOutput LevelsImpl(PositionInput position, Params0Input params1, Params1Input params2) {
  Consts0Input consts0 = Consts0Input{6.28, 3.14};
  Consts1Input consts1 = Consts1Input{float2{2.0, 2.0}, float2{1.0, 1.0}};
  Consts2Input consts2 = Consts2Input{float2{1.0, 1.0}};
  Consts3Input consts3 = Consts3Input{0.0, 1.0};
  ScaleOut scale0 = ScaleImpl(ScaleIn{position.position, params1.scale});
  NormalizeOut normalize0 = NormalizeImpl(NormalizeIn{params1.theta, consts0.double_pi, consts0.pi});
  Normalize2fOut normalize2f0 = Normalize2fImpl(Normalize2fIn{params1.trans, consts1.two, consts1.one});
  RotationOut rotation0 = RotationImpl(RotationIn{scale0.position, normalize0.value});
  TranslationOut translation0 = TranslationImpl(TranslationIn{rotation0.position, normalize2f0.value});
  RepetitionOut repetition0 = RepetitionImpl(RepetitionIn{translation0.position, consts2.space});
  GridOut grid0 = GridImpl(GridIn{repetition0.position, params1.thickness});
  MulOut mul0 = MulImpl(MulIn{grid0.distance, params1.scale});
  Level1Output level1 = Level1Output{mul0.value, consts3.levelid0};
  TruchetOut truchet0 = TruchetImpl(TruchetIn{translation0.position, params2.thickness, params2.rotation});
  MulOut mul1 = MulImpl(MulIn{truchet0.distance, params1.scale});
  ScaffoldIdOut scaffoldid0 = ScaffoldIdImpl(ScaffoldIdIn{mul0.value, consts3.levelid0, mul1.value, consts3.levelid1, position.view_scaffold});
  Level2Output level2 = Level2Output{scaffoldid0.distance, scaffoldid0.levelid};
  return LevelsOutput{level1, level2};
}

